
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__  void  AplusB( int *sum,  int *a,  int *b, int n) {
/* 
 * Return the sum of the `a` and `b` arrays
 */
    // Fetch the index
    int i = blockIdx.x;
    // Perform the sum
    sum[i] = a[i] + b[i];
    
} // ---


int main() {
/*
 * Calculate the sum of two vectors using non-managed memory
 */
    int n = 1000;
    
    // <-- HOST memory management
    
        // Create the vectors in the HOST
        int *sum, *a, *b;
        
        sum = (int *) malloc( n * sizeof(int) );
        a = (int *) malloc( n * sizeof(int) );
        b = (int *) malloc( n * sizeof(int) );
        
        // Fill the vectors in the host
        for( int i=0; i<n; i++) {
            
            a[i] = i*i + i;
            b[i] = -i*i; // a[i]+b[i] = i
        }
        
    // <-- DEVICE memory management
    
        // Create the vectors in the DEVICE
        int *d_sum, *d_a, *d_b;
        
        hipMalloc(&d_sum, n * sizeof(int));
        hipMalloc(&d_a, n * sizeof(int));
        hipMalloc(&d_b, n * sizeof(int));
        
        // Copy the vectors to the DEVICE
        hipMemcpy( d_a, a, n * sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( d_b, b, n * sizeof(int), hipMemcpyHostToDevice );
        
    // <-- Main calculation
    
        AplusB<<< n, 1 >>>(d_sum, d_a, d_b, n);
        
        // Get the sum vector from the device
        hipMemcpy(sum, d_sum, n * sizeof(int), hipMemcpyDeviceToHost);
        
        //Free unneeded memory
        hipFree(d_sum);
        hipFree(d_a);
        hipFree(d_b);
        
    
    // <-- Display results
        
        // Display the results
        for(int i=0; i<n; i++) {
            printf("%d: %d + %d = %d\n", i, a[i], b[i], sum[i]); 
        }
        
        // Free unneeded memory
        free(sum);
        free(a);
        free(b);
        
    return  0;
    
} // ---
