#include "hip/hip_runtime.h"

__global__  void  AplusB( int *sum,  int *a,  int *b, int n) {
/* 
 * Return the sum of the `a` and `b` arrays
 */
    // Fetch the index
    int i = threadIdx.x;
    // Perform the sum
    ret[i] = a[i] + b[i];
    
} // ---


int main() {
/*
 * Calculate the sum of two vectors using non-managed memory
 */
    int n = 1000;
    
    // <-- HOST memory management
    
        // Create the vectors in the HOST
        int *sum, *a, *b;
        
        sum = malloc( n * sizeof(int) );
        a = malloc( n * sizeof(int) );
        b = malloc( n * sizeof(int) );
        
        // Fill the vectors in the host
        for( int i=0; i<n; i++) {
            
            a[i] = i*i + i;
            b[i] = -i*i; // a[i]+b[i] = i
        }
        
    // <-- DEVICE memory management
    
        // Create the vectors in the DEVICE
        int *d_sum, *d_a, *d_b;
        
        hipMalloc(&d_sum, n * sizeof(int));
        hipMalloc(&d_a, n * sizeof(int));
        hipMalloc(&d_b, n * sizeof(int));
        
        // Copy the vectors to the DEVICE
        hipMemcpy( d_a, a, n * sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( d_b, b, n * sizeof(int), hipMemcpyHostToDevice );
        
    // <-- Main calculation
    
        AplusB<<< n, 1 >>>(d_sum, d_a, d_b, n);
        
        // Get the sum vector from the device
        hipMemcpy(sum, d_sum, n * sizeof(int), hipMemcpyDeviceToHost);
        
        //Free unneeded memory
        hipFree(d_sum);
        hipFree(d_a);
        hipFree(d_b);
        
    
    // <-- Display results
        
        // Display the results
        for(int i=0; i<n; i++) {
            printf("%d: %d + %d = %d\n", i, a[i], b[i], sum[i]); 
        }
        
        // Free unneeded memory
        free(sum);
        free(a);
        free(b);
        
    return  0;
    
} // ---
