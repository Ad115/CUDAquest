
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__  void  AplusB( int  *ret,  int  a,  int  b) {
/*
 * Simple unimportant kernel
 */
    ret[threadIdx.x] = a + b + threadIdx.x;
}


int main() {

    // Create space in the device
    int *ret;
    hipMalloc(&ret, 1000 * sizeof(int));

    // Call the kernel
    AplusB<<< 1, 1000 >>>(ret, 10, 100);

    // Recieve the results
    int *host_ret = (int *)malloc(1000 * sizeof(int));
    hipMemcpy(host_ret, ret, 1000 * sizeof(int), hipMemcpyDefault);

    // Display the results
    for(int i=0; i<1000; i++) {
        printf("%d: A+B = %d\n", i, host_ret[i]); 
    }

    // Free unneeded memory
    free(host_ret);
    hipFree(ret); 

    return  0;
}
