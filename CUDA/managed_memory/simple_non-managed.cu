
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__  void  AplusB( int  *ret,  int  a,  int  b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}
int main() {
    int *ret;
    hipMalloc(&ret, 1000 * sizeof(int));
    AplusB<<< 1, 1000 >>>(ret, 10, 100);
    int *host_ret = (int *)malloc(1000 * sizeof(int));
    hipMemcpy(host_ret, ret, 1000 * sizeof(int), hipMemcpyDefault);
    for(int i=0; i<1000; i++)
        printf("%d: A+B = %d\n", i, host_ret[i]); 
    free(host_ret);
    hipFree(ret); 
    return  0;
}


