#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c = -10;
    int *gpu_c;

    // Create memory on device
    int errorMalloc = hipMalloc( &gpu_c, sizeof(int) );
    // Call add function on device
    add<<<1,1>>>(2, 3, gpu_c);
    // Copy the result back to the host
    int errorMemcpy = hipMemcpy( &c, gpu_c, sizeof(int), hipMemcpyDeviceToHost );
    // Free space used on device
    hipFree(gpu_c);

    printf("2 + 3 = %d \n", c);
	
	printf("Errors found: hipMalloc -> %s, hipMemcpy -> %s)\n", 
		   hipGetErrorString( errorMalloc ),
		   hipGetErrorString( errorMemcpy )
	);

    return 0;
}
