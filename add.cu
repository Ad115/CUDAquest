
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c = -10;
    int *gpu_c;

    // Create memory on device
    hipError_t errorMalloc = hipMalloc( &gpu_c, sizeof(int) );
    // Call add function on device
    add<<<1,1>>>(2, 3, gpu_c);
    // Copy the result back to the host
    hipError_t errorMemcpy = hipMemcpy( &c, gpu_c, sizeof(int), hipMemcpyDeviceToHost );
    // Free space used on device
    hipFree(gpu_c);

    printf("2 + 3 = %d \n", c);
	
	printf("Errors found: \n\thipMalloc -> %s, \n\thipMemcpy -> %s\n", 
		   hipGetErrorString( errorMalloc ),
		   hipGetErrorString( errorMemcpy )
	);

    return 0;
}
